
#include <hip/hip_runtime.h>
// __global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

//     // Get the row and column indices of the matrix C element being processed
//     int row = blockIdx.y*blockDim.y + threadIdx.y;
//     int col = blockIdx.x*blockDim.x + threadIdx.x;

//     // Check if the indices are within the bounds of the matrix C
//     if (row < Ni && col < Nj)
//     {
//         double value = 0;
//         for (int k = 0; k < Nk; k++)
//         {
//             //value += A[row*Nk + k] * B[k*Nj + col];
//             value += A[k*Ni + row] * B[k*Nj + col];
//         }
//         C[row*Nj + col] = value;
//         //C[col*Ni+row] = value;
//     }
// }


__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk)
{   int BLOCK_SIZE = 8;
    __shared__ double mat_1_tile[8][8];
    __shared__ double mat_2_tile[8][8];

    double acc_sum{0};
    int temp = ceil((double)Nk / BLOCK_SIZE);
    for (int tile_idx = 0; tile_idx < temp; ++tile_idx)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = tile_idx * blockDim.x + threadIdx.x;
        if ((i < Ni) && (j < Nk))
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = A[j * Ni + i];
            //mat_1_tile[threadIdx.x][threadIdx.y] = A[i * Nk + j];
        }
        else
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = 0;
        }
        i = tile_idx * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if ((i < Nk) && (j < Nj))
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = B[i * Nj + j];
        }
        else
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            acc_sum += mat_1_tile[threadIdx.y][k] * mat_2_tile[k][threadIdx.x];
        }
        __syncthreads();
    }

    // 2D block and 2D thread
    // Each thread computes one cell in C.
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i < Ni) && (j < Nj))
    {
        C[i * Nj + j] = acc_sum;
    }
}
