
#include <hip/hip_runtime.h>

// matrix multiply kernel: C = A^T * B^T

#define BLOCK_SIZE 8
/******************
Base Version
***********/
// __global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

//     // Get the row and column indices of the matrix C element being processed
//     int row = blockIdx.y*blockDim.y + threadIdx.y;
//     int col = blockIdx.x*blockDim.x + threadIdx.x;

//     // Check if the indices are within the bounds of the matrix C
//     if (row < Ni && col < Nj)
//     {
//         double value = 0;
//         for (int k = 0; k < Nk; k++)
//         {
//             //value += A[row*Nk + k] * B[k*Nj + col];
//             value += A[k*Ni + row] * B[col*Nk + k];
//         }
//         C[row*Nj + col] = value;
//         //C[col*Ni+row] = value;
//     }
// }

/******************
Shared memory version 
***********/

__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk)
{   //int BLOCK_SIZE = 16;
    __shared__ double mat_1_tile[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double mat_2_tile[BLOCK_SIZE][BLOCK_SIZE];

    double acc_sum{0};
    int temp = ceil((double)Nk / BLOCK_SIZE);
    for (int tile_idx = 0; tile_idx < temp; ++tile_idx)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = tile_idx * blockDim.x + threadIdx.x;
        if ((i < Ni) && (j < Nk))
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = A[j * Ni + i];
            //mat_1_tile[threadIdx.x][threadIdx.y] = A[i * Nk + j];
        }
        else
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = 0;
        }
        i = tile_idx * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if ((i < Nk) && (j < Nj))
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = B[j * Nk + i];
        }
        else
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            acc_sum += mat_1_tile[threadIdx.y][k] * mat_2_tile[k][threadIdx.x];
        }
        __syncthreads();
    }

    // 2D block and 2D thread
    // Each thread computes one cell in C.
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i < Ni) && (j < Nj))
    {
        C[i * Nj + j] = acc_sum;
    }
}


/******************
Unrolling-k loop
***********/

// __global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

//     // Get the row and column indices of the matrix C element being processed
//     int row = (blockIdx.y*blockDim.y + threadIdx.y);
//     int col = (blockIdx.x*blockDim.x + threadIdx.x);

//     // Check if the indices are within the bounds of the matrix C
//     if (row < Ni && col < Nj)
//     {
//         double value = 0;
//         for (int k = 0; k < Nk; k+=2)
//         {
//             //value += A[row*Nk + k] * B[k*Nj + col];
//             value += A[k*Ni + row] * B[col*Nk + k];
//             value += A[(k+1)*Ni + row] * B[col*Nk + k+1];
            
//         }
//         C[row*Nj + col] = value;
        
//         //C[col*Ni+row] = value;
//     }
// }




